#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */
#include <iostream>
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include ""
// #define __HIPCC__
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "simple_knn.h"
#include "auxiliary.h"

struct CustomMin
{
	__device__ __forceinline__ float3 operator()(const float3 &a, const float3 &b) const
	{
		return {min(a.x, b.x), min(a.y, b.y), min(a.z, b.z)};
	}
};

struct CustomMax
{
	__device__ __forceinline__ float3 operator()(const float3 &a, const float3 &b) const
	{
		return {max(a.x, b.x), max(a.y, b.y), max(a.z, b.z)};
	}
};

__device__ __forceinline__ float dist2(const float3 &a, const float3 &b)
{
	float3 d = {a.x - b.x, a.y - b.y, a.z - b.z};
	return d.x * d.x + d.y * d.y + d.z * d.z;
}

__host__ __device__ uint32_t prepMorton(uint32_t x)
{
	x = (x | (x << 16)) & 0x030000FF;
	x = (x | (x << 8)) & 0x0300F00F;
	x = (x | (x << 4)) & 0x030C30C3;
	x = (x | (x << 2)) & 0x09249249;
	return x;
}

__host__ __device__ uint32_t coord2Morton(const float3 &coord, const float3 &minn, const float3 &maxx)
{
	uint32_t x = prepMorton(((coord.x - minn.x) / (maxx.x - minn.x)) * ((1 << 10) - 1));
	uint32_t y = prepMorton(((coord.y - minn.y) / (maxx.y - minn.y)) * ((1 << 10) - 1));
	uint32_t z = prepMorton(((coord.z - minn.z) / (maxx.z - minn.z)) * ((1 << 10) - 1));

	return x | (y << 1) | (z << 2);
}

__global__ void coord2Morton(int P, const float3 *points, float3 minn, float3 maxx, uint32_t *codes)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	codes[idx] = coord2Morton(points[idx], minn, maxx);
}

struct MinMax
{
	float3 minn;
	float3 maxx;
};

__global__ void boxMinMax(int P, const float3 *points, const uint32_t *indices, MinMax *boxes)
{
	auto idx = cg::this_grid().thread_rank();

	MinMax me;
	if (idx < P)
	{
		me.minn = points[indices[idx]];
		me.maxx = points[indices[idx]];
	}
	else
	{
		me.minn = {FLT_MAX, FLT_MAX, FLT_MAX};
		me.maxx = {-FLT_MAX, -FLT_MAX, -FLT_MAX};
	}

	__shared__ MinMax redResult[BOX_SIZE];

	for (int off = BOX_SIZE / 2; off >= 1; off /= 2)
	{
		if (threadIdx.x < 2 * off)
			redResult[threadIdx.x] = me;
		__syncthreads();

		if (threadIdx.x < off)
		{
			MinMax other = redResult[threadIdx.x + off];
			me.minn.x = min(me.minn.x, other.minn.x);
			me.minn.y = min(me.minn.y, other.minn.y);
			me.minn.z = min(me.minn.z, other.minn.z);
			me.maxx.x = max(me.maxx.x, other.maxx.x);
			me.maxx.y = max(me.maxx.y, other.maxx.y);
			me.maxx.z = max(me.maxx.z, other.maxx.z);
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
		boxes[blockIdx.x] = me;
}

__device__ __host__ float distBoxPoint(const MinMax &box, const float3 &p)
{
	float3 diff = {0, 0, 0};
	if (p.x < box.minn.x || p.x > box.maxx.x)
		diff.x = min(abs(p.x - box.minn.x), abs(p.x - box.maxx.x));
	if (p.y < box.minn.y || p.y > box.maxx.y)
		diff.y = min(abs(p.y - box.minn.y), abs(p.y - box.maxx.y));
	if (p.z < box.minn.z || p.z > box.maxx.z)
		diff.z = min(abs(p.z - box.minn.z), abs(p.z - box.maxx.z));
	return diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
}

template <int K>
__device__ void updateKBest(const float3 &ref, const float3 &point, float *knn)
{
	float dist = dist2(ref, point);
	for (int j = 0; j < K; j++)
	{
		if (knn[j] > dist)
		{
			float t = knn[j];
			knn[j] = dist;
			dist = t;
		}
	}
}

__global__ void boxMeanDist(int P, const float3 *points, const uint32_t *indices, const MinMax *boxes, float *dists)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 point = points[indices[idx]];
	float best[3] = {FLT_MAX, FLT_MAX, FLT_MAX};

	for (int i = max(0, idx - 3); i <= min(P - 1, idx + 3); i++)
	{
		if (i == idx)
			continue;
		updateKBest<3>(point, points[indices[i]], best);
	}

	float reject = best[2];
	best[0] = FLT_MAX;
	best[1] = FLT_MAX;
	best[2] = FLT_MAX;

	for (int b = 0; b < (P + BOX_SIZE - 1) / BOX_SIZE; b++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist > reject || dist > best[2])
			continue;

		for (int i = b * BOX_SIZE; i < min(P, (b + 1) * BOX_SIZE); i++)
		{
			if (i == idx)
				continue;
			updateKBest<3>(point, points[indices[i]], best);
		}
	}
	dists[indices[idx]] = (best[0] + best[1] + best[2]) / 3.0f;
}

__global__ void boxNearestNeighbor(int P, int batch_size, const float3 *points, const uint32_t *indices, const MinMax *boxes, uint32_t *indices_nearest)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	uint32_t c_idx = indices[idx];
	float3 point = points[c_idx];
	float best = FLT_MAX;
	float dist;
	uint32_t nearest, ii;

	for (int i = max(0, idx - 3); i <= min(P - 1, idx + 3); i++)
	{
		ii = indices[i];
		if (ii / batch_size == c_idx / batch_size)
			continue;
		dist = dist2(point, points[ii]);
		if (dist < best)
		{
			best = dist;
			nearest = ii;
		}
	}

	float reject = best;
	best = FLT_MAX;

	for (int b = 0; b < (P + BOX_SIZE - 1) / BOX_SIZE; b++)
	{
		MinMax box = boxes[b];
		dist = distBoxPoint(box, point);
		if (dist > reject || dist > best)
			continue;

		for (int i = b * BOX_SIZE; i < min(P, (b + 1) * BOX_SIZE); i++)
		{
			ii = indices[i];
			if (ii / batch_size == c_idx / batch_size)
				continue;
			dist = dist2(point, points[ii]);
			if (dist < best)
			{
				best = dist;
				nearest = ii;
			}
		}
	}

	indices_nearest[c_idx] = nearest;
}

void SimpleKNN::knn(int P, const float3 *points, float *meanDists)
{
	size_t temp_storage_bytes;
	float3 init = {0, 0, 0}, minn, maxx;

	// calculate points bounding box
	float3 *result;
	hipMalloc(&result, sizeof(float3));
	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);
	CHECK_CUDA();

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);
	hipFree(result);
	CHECK_CUDA();

	// calculate morton codes
	thrust::device_vector<uint32_t> morton(P);
	coord2Morton<<<(P + 255) / 256, 256>>>(P, points, minn, maxx, morton.data().get());
	CHECK_CUDA();

	// sort morton codes
	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);
	thrust::device_vector<uint32_t> morton_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	// calculate subgroups bounding boxes
	uint32_t num_boxes = (P + BOX_SIZE - 1) / BOX_SIZE;
	thrust::device_vector<MinMax> boxes(num_boxes);
	boxMinMax<<<num_boxes, BOX_SIZE>>>(P, points, indices_sorted.data().get(), boxes.data().get());
	CHECK_CUDA();

	// calculate knn distances
	boxMeanDist<<<num_boxes, BOX_SIZE>>>(P, points, indices_sorted.data().get(), boxes.data().get(), meanDists);
	CHECK_CUDA();
}

void SimpleKNN::nearestNeighbor(int P, int batch_size, const float3 *points, uint32_t *indices_nearest)
{
	size_t temp_storage_bytes;
	float3 init = {0, 0, 0}, minn, maxx;

	// calculate points bounding box
	float3 *result;
	hipMalloc(&result, sizeof(float3));
	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);
	CHECK_CUDA();

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);
	hipFree(result);
	CHECK_CUDA();

	// calculate morton codes
	thrust::device_vector<uint32_t> morton(P);
	coord2Morton<<<(P + 255) / 256, 256>>>(P, points, minn, maxx, morton.data().get());
	CHECK_CUDA();

	// sort morton codes
	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);
	thrust::device_vector<uint32_t> morton_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	// calculate subgroups bounding boxes
	uint32_t num_boxes = (P + BOX_SIZE - 1) / BOX_SIZE;
	thrust::device_vector<MinMax> boxes(num_boxes);
	boxMinMax<<<num_boxes, BOX_SIZE>>>(P, points, indices_sorted.data().get(), boxes.data().get());
	CHECK_CUDA();

	// calculate nearest neighbors
	boxNearestNeighbor<<<num_boxes, BOX_SIZE>>>(P, batch_size, points, indices_sorted.data().get(), boxes.data().get(), indices_nearest);
	CHECK_CUDA();
}
